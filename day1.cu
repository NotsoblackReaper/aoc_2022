#include "hip/hip_runtime.h"
#include "AOC_Methods.cuh"

	uint64_t aoc::day1::part_1(int* data, int length) {
		thrust::host_vector<int>tmp(1);
		thrust::host_vector<int>elfs(1);
		for (int i = 0; i < length; ++i) {
			if (data[i] == 0) {
				elfs.push_back(thrust::reduce(tmp.begin(), tmp.end()));
				tmp.clear();
				continue;
			}
			tmp.push_back(data[i]);
		}
		auto max = *(thrust::max_element(elfs.begin(), elfs.end()));
		return max;
	}

	uint64_t aoc::day1::part_2(int* data, int length) {
		thrust::host_vector<int>tmp(1);
		thrust::host_vector<int>elfs(1);
		for (int i = 0; i < length; ++i) {
			if (data[i] == 0) {
				elfs.push_back(thrust::reduce(tmp.begin(), tmp.end()));
				tmp.clear();
				continue;
			}
			tmp.push_back(data[i]);
		}

		thrust::sort(elfs.begin(), elfs.end());
		size_t size = elfs.size();
		int max = elfs.back() + elfs[size - 2] + elfs[size - 3];
		return max;
	}